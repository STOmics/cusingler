#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include "cusingler.cuh"

#include <stdlib.h>
#include <stdio.h>

#include <cmath>
#include <chrono>
#include <thread>
#include <iostream>
#include <set>
#include <algorithm>
#include "hip/hip_runtime.h"
#include "hip/hip_math_constants.h"
#define LOG
hipError_t errcode;
hipStream_t stream;
float* d_ref, *d_qry;
vector<float> h_labels;
uint32 ref_height, ref_width, qry_height, qry_width;
uint32 ct_num;
uint32* d_ctids;
vector<uint32> h_ctidx;
vector<uint32> h_ctdiff, h_ctdidx;
size_t pitchref;
size_t pitchqry;

uint32* d_gene_idx;
float* d_qry_line, *d_qry_rank;
int* h_qry_idx_sample;//  to be realesed
int* d_qry_idx_sample;
int* d_qry_idx;    //idx for pair sort
float* d_ref_lines, *d_ref_rank;
float *d_score;

// unit is MB
#define CHECK(call)                                                       \
{                                                                         \
   const hipError_t error = call;                                        \
   if (error != hipSuccess)                                              \
   {                                                                      \
      printf("Error: %s:%d, ", __FILE__, __LINE__);                       \
      printf("code:%d, reason: %s\n", error, hipGetErrorString(error));  \
      exit(1);                                                            \
   }                                                                      \
}

uint32 getUsedMem()
{
    size_t free, total;
    hipMemGetInfo(&free, &total);
    return (total-free)/1024/1024;
}



bool init()
{
    stream =NULL;
    d_ref = NULL;
    d_qry = NULL;
    ref_height = ref_width = qry_height = qry_width = 0;
    ct_num = 0;
    d_ctids = NULL;
    pitchref=0;
    pitchqry=0;
    return true;
}

bool destroy()
{
    hipFree(d_ref);
    hipFree(d_qry);
    // hipFree(d_labels);
    hipFree(d_ctids);
    hipStreamDestroy(stream);
    // hipFree(d_ctidx);
    // hipFree(d_ctdiff);
    // hipFree(d_ctdidx);
    free(h_qry_idx_sample);


    hipFree(d_gene_idx);
    hipFree(d_qry_line);
    hipFree(d_qry_rank);
    hipFree(d_ref_lines);
    hipFree(d_ref_rank);
    hipFree(d_score);

    return true;
}


bool rank_by_unique(float* dataInOut ,int* dataIdx,int datalen)
{
    //thrust::device_ptr<float> thrust_InOut(dataInOut);
    // for(int i=0;i<datalen;i++)
    // {
    //     cout<<"data: "<<thrust_InOut[i]<<endl;
    // }
    thrust::device_vector<float> datavec(dataInOut,dataInOut+datalen);
    thrust::device_vector<float> out(datalen) ;
    thrust::sort(datavec.begin(),datavec.end());
    // for(int i=0;i<datalen;i++)
    // {
    //     cout<<"data: "<<datavec[i]<<endl;
    // }
    
    auto eend= thrust::unique_copy(datavec.begin(), datavec.end(), out.begin());
    auto num_unique = thrust::distance(out.begin(), eend);

     cout<<"uniq cnt"<<num_unique<<endl;
     for(int i=0;i<num_unique;i++)
     {
         cout<<"uniq data: "<<out[i]<<endl;
     }
    return true;
}


__global__ void sortrank(float* dataInOut, int* dataIdx,int datalen)
{   
    uint32 tid=blockIdx.x*blockDim.x+threadIdx.x;
    uint32 tid_idx;
    uint32 offset=0;
    uint32 num_swaps;
    uint32 tid_idx_max=datalen-1;
do 
{
    num_swaps=0;
    tid_idx=tid*2+offset;
    if(tid_idx<tid_idx_max)
    {
       
        if(dataInOut[tid_idx]>dataInOut[tid_idx+1])
        {
            float tmp=dataInOut[tid_idx];
            dataInOut[tid_idx]=dataInOut[tid_idx+1];
            dataInOut[tid_idx+1]=tmp;
            int tmpidx=dataIdx[tid_idx];
            dataIdx[tid_idx]=dataIdx[tid_idx+1];
            dataIdx[tid_idx+1]=tmpidx;
            num_swaps=1;
        }
        
    }
    offset=1-offset;
}while(__syncthreads_count(num_swaps)!=0);

}
//get rank after sort
__global__ void sumrank(float* dataInOut,int* rankidx,const int datalen)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
   // __shared__ float s_rank[1024] ;
    if(tid<datalen)
    {
        int idxL=tid;
        int idxR=tid;
        while(idxL>0&(dataInOut[idxL]==dataInOut[idxL-1]))
            idxL-=1;
        do
        {
            idxR+=1;
        }while (idxR!=datalen&(dataInOut[idxR]==dataInOut[idxR-1]));
     //   s_rank[threadIdx.x]=1+idxL+float(idxR-idxL-1)/2;
      float tmp=1+idxL+float(idxR-idxL-1)/2;
        __syncthreads();
    //    dataInOut[rankidx[tid]]=s_rank[threadIdx.x];
    dataInOut[rankidx[tid]]=tmp;
    }

}
bool sort_by_idx1(float* dataInOut ,int* dataIdx,int datalen)
{
    //get thrust ptr
    // std::cout<<"used gpu mem(MB): "<<getUsedMem()<<std::endl;
    thrust::device_ptr<float> thrust_InOut(dataInOut);
    thrust::device_ptr<int> thrust_Idx(dataIdx);     
    // getchar();
    // std::cout<<"used gpu mem(MB): "<<getUsedMem()<<std::endl;  
    // getchar();
    
    //
    // thrust::sort_by_key(thrust_InOut,thrust_InOut+datalen,thrust_Idx);
    // for(int i=0;i<datalen;i++)
    // {
    //     cout<<"data: "<<thrust_InOut[i]<<"idx："<<thrust_Idx[i]<<endl;
    // }
    //get rank
    sumrank<<<datalen/256+1,256>>>(dataInOut,dataIdx,datalen);
    // for(int i=0;i<datalen;i++)
    // {
    //     cout<<"data: "<<thrust_InOut[i]<<"idx："<<thrust_Idx[i]<<endl;
    // }
    // getchar();
    return true;
}
bool sort_by_idx(float* dataInOut ,int* dataIdx,int *uniqueidx,float*uniqueout,int datalen)
{
    //get thrust ptr
    thrust::device_ptr<float> thrust_InOut(dataInOut);
    // thrust::device_ptr<float> thrust_Out(dataOut);
    // create index on dev
//     thrust::device_ptr<int> thrust_Idx(dataIdx);  
//     thrust::device_vector<float> uniqueout(datalen);
//     thrust::device_vector<int> uniqueidx(datalen);
//     //sort
//    // cout<<"length rank"<<datalen<<endl;
//     //thrust::unique
//     thrust::sort_by_key(thrust_InOut,thrust_InOut+datalen,thrust_Idx);
//     auto eend=thrust::unique_copy(thrust_InOut,thrust_InOut+datalen,uniqueout);
//     // for(int i=0;i<datalen;i++)
//     // {
//     //     cout<<"data: "<<thrust_InOut[i]<<" idx: "<<thrust_Idx[i]<<endl;
//     // }
//     auto num_unique = thrust::distance(uniqueout, eend);
    //for test
    //uniqueout.data();
    //uniqueidx.data();

    #ifdef LOG
    // for(int i=0;i<num_unique;i++)
    // {
    // cout<<"datauni: "<<uniqueout[i]<<endl;
    // }
    //getchar();
    #endif


    return true;
}



bool copyin(InputData& rawdata, vector<uint32>& ctids, vector<uint32>& ctidx, vector<uint32>& ctdiff, vector<uint32>& ctdidx)
{
    

    ref_height = rawdata.ref_cell_num;
    ref_width = rawdata.ref_gene_num;
    qry_height = rawdata.test_cell_num;
    qry_width = rawdata.test_gene_num;
    ct_num = rawdata.celltypes.size();

    // float max_val = 0;
    // for (int i = 0; i < qry_width; ++i)
    // {
    //     max_val = max(max_val, rawdata.test[i]);
    // }
    // cout<<"qry width: "<<qry_width<<endl;
    // cout<<"qry max value: "<<max_val<<endl;
    //
    CHECK(hipStreamCreate(&stream));
    
    CHECK(hipMallocPitch((void**)&d_ref,&pitchref,ref_width*sizeof(float),ref_height));
    CHECK(hipMallocPitch((void**)&d_qry,&pitchqry,qry_width*sizeof(float),qry_height));
    
    std::cout<<"pitchref: "<<pitchref<<std::endl;
    std::cout<<"ref_width: "<<ref_width<<std::endl;
   // std::cout<<"pitchqry: "<<pitchqry<<std::endl;

    //hipMalloc((void**)&d_ref, ref_height * ref_width * sizeof(float));
    //hipMalloc((void**)&d_qry, qry_height * qry_width * sizeof(float));
    // hipMalloc((void**)&d_labels, qry_height * ct_num * sizeof(float));
    hipMalloc((void**)&d_ctids, ctids.size() * sizeof(uint32));
    // hipMalloc((void**)&d_ctidx, ctidx.size() * sizeof(uint32));
    // hipMalloc((void**)&d_ctdiff, ctdiff.size() * sizeof(uint32));
    // hipMalloc((void**)&d_ctdidx, ctdidx.size() * sizeof(uint32));
    cout<<"current stream"<<stream<<endl;
    hipMemcpy2DAsync(d_ref,pitchref, rawdata.ref.data(), ref_width * sizeof(float),ref_width * sizeof(float),ref_height,hipMemcpyHostToDevice,stream);
    hipMemcpy2DAsync(d_qry,pitchqry, rawdata.test.data(),qry_width * sizeof(float),qry_width * sizeof(float),qry_height,hipMemcpyHostToDevice,stream);

    // hipMemcpyAsync(d_ref, rawdata.ref.data(), ref_height * ref_width * sizeof(float), hipMemcpyHostToDevice,stream);
    // hipMemcpyAsync(d_qry, rawdata.test.data(), qry_height * qry_width * sizeof(float), hipMemcpyHostToDevice,stream);
    // // hipMemcpy(d_labels, rawdata.labels.data(), qry_height * ct_num * sizeof(float), hipMemcpyHostToDevice);
    h_labels = rawdata.labels;
    //CHECK( hipMemcpyAsync(d_ctids, ctids.data(), ctids.size() * sizeof(uint32), hipMemcpyHostToDevice,stream));
    CHECK(hipMemcpy(d_ctids, ctids.data(), ctids.size() * sizeof(uint32), hipMemcpyHostToDevice));
   
    // hipMemcpy(d_ctidx, ctidx.data(), ctidx.size() * sizeof(uint32), hipMemcpyHostToDevice);
    h_ctidx = ctidx;
    // hipMemcpy(d_ctdiff, ctdiff.data(), ctdiff.size() * sizeof(uint32), hipMemcpyHostToDevice);
    // hipMemcpy(d_ctdidx, ctdidx.data(), ctdidx.size() * sizeof(uint32), hipMemcpyHostToDevice);
    h_ctdiff = ctdiff;
    h_ctdidx = ctdidx;
    hipStreamSynchronize(stream);
    // std::this_thread::sleep_for(std::chrono::seconds(5));
    std::cout<<"used gpu mem(MB): "<<getUsedMem()<<std::endl;

    hipMalloc((void**)&d_gene_idx, qry_width * sizeof(uint32));
    hipMalloc((void**)&d_qry_line, qry_width * sizeof(float));
    hipMalloc((void**)&d_qry_rank, qry_width * sizeof(float));
    int idx_len=10000;
    hipMalloc((void**)&d_qry_idx_sample,idx_len*sizeof(int));
    //create origin idx array on CPU and copy to GPU
    h_qry_idx_sample=(int*)malloc(idx_len*sizeof(idx_len));
    for (int i = 0; i < idx_len; ++i)
    {
        h_qry_idx_sample[i] = i;

    }
    CHECK(hipMemcpyAsync(d_qry_idx_sample,h_qry_idx_sample,idx_len*sizeof(int),hipMemcpyHostToDevice,stream));
    //use idx sample to reset idx array on GPU
    hipMalloc((void**)&d_qry_idx,idx_len*sizeof(int));
    hipMalloc((void**)&d_ref_lines, 100000000 * sizeof(float));
    hipMalloc((void**)&d_ref_rank, 100000000 * sizeof(float));
    hipMalloc((void**)&d_score, 100000 * sizeof(float));

    return true;
}



__global__ void get_device_qry_line(uint32* gene_idx, float* qry, const uint32 len, const uint32 gene_len, float* res)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < len)
    {
        res[tid] = qry[gene_len-gene_idx[tid]-1];//gene_len-1=idx_max    g_idx  int->float res dqry-line descending order
    }
}

__global__ void get_device_ref_lines(uint32* gene_idx, const uint32 gene_len,
    uint32* cell_idx, const uint32 cell_len, float* ref, const uint32 ref_width, 
    const uint32 ref_pitch, float* res)
{
    int nx = blockIdx.x * blockDim.x + threadIdx.x;
    int ny = blockIdx.y * blockDim.y + threadIdx.y;
    if (nx < cell_len && ny < gene_len)
    {
        float* row_head = (float*)((char*)ref + (uint64)(cell_idx[nx]) * ref_pitch);
        res[nx * gene_len + ny] = row_head[ref_width - gene_idx[ny] - 1];
    }
}

__global__ void rankdata(float* qry, const uint32 len, float* res)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < len)
    {
        float r = 1, s = 0;
        for (int i = 0; i < len; ++i)
        {
            if (qry[tid] == qry[i])
                s += 1;
            else if (qry[tid] > qry[i])
                r += 1;
        }
        res[tid] = r + float(s-1)/2;
    }
}

__global__ void spearman(float* qry, float* ref, const uint32 gene_num, const uint32 cell_num, float* score)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < cell_num)
    {
        float mean = (gene_num+1)/2.0;
        float sumxy = 0, sumxx = 0, sumyy = 0;
        for (int i = 0; i < gene_num; ++i)
        {
            float x = qry[i] - mean;
            float y = ref[tid * gene_num + i] - mean;
            sumxy += x * y;
            sumxx += x * x;
            sumyy += y * y;
        }
        float divisor = sqrt(sumxx * sumyy);
        if (divisor != 0)
            score[tid] = sumxy / divisor;
        else
            score[tid] = HIP_NAN_F;
    }
}

float percentile(vector<float> arr, int len, float p)
{
    if (len <= 1) return arr.front();

    float res;
    std::sort(arr.begin(), arr.begin()+len);

    vector<float> index;
    float step = 1.0/(len-1);
    for (int i = 0; i < len; ++i)
        index.push_back(i*step);
   
    if (p <= index.front())
    {
        res = arr[0];
    }
    else if (index.back() <= p)
    {
        res = arr[len-1];
    }
    else
    {
        auto it = lower_bound(index.begin(), index.end(), p);
        float prevIndex = *(it-1);
        float prevValue = arr.at(it - index.begin()-1);
        float nextValue = arr.at(it - index.begin());
        // linear interpolation
        res = (p - prevIndex) * (nextValue - prevValue) / step + prevValue;
    }
    return res;
}

vector<uint32> finetune_round(float* qry, vector<uint32> top_labels)
{
    clock_t t0,t1;
    float time;
    // get filtered genes
    // cout<<"top_labels: ";
    // for (auto& label : top_labels)
    //     cout<<label<<" ";
    // cout<<"\ntop label num: "<<top_labels.size() <<" ct_num: "<<ct_num<<endl;
    set<uint32> uniq_genes;
    int gene_thre = round(500 * pow((2/3.0), log2(top_labels.size())));
    // cout<<"gene_thre: "<<gene_thre<<endl;
    t0=clock();
    
    for (auto& i : top_labels)//??line 159  topl cant be 0??
    {
        for (auto& j : top_labels)
        {
            if (i == j)//same cant be 0?
                continue;
            int pos = h_ctdidx[(i * ct_num + j) * 2];
            int len = h_ctdidx[(i * ct_num + j) * 2 + 1];
            if (len > gene_thre)
                len = gene_thre;
            uniq_genes.insert(h_ctdiff.begin()+pos, h_ctdiff.begin()+pos+len);
            // cout<<"temp uniq genes size: "<<uniq_genes.size()<<endl;
        }
    }
    t1=clock();
    time=(float)(t1-t0)/CLOCKS_PER_SEC;
    cout<<"get uniq_genes: "<<time<<endl;
    // cout<<"uniq genes size: "<<uniq_genes.size()<<endl;
    // t0=clock();
    vector<uint32> h_gene_idx(uniq_genes.begin(), uniq_genes.end());
    
    // transfer qry data from cpu to gpu
    CHECK(hipMemcpy(d_gene_idx, h_gene_idx.data(), h_gene_idx.size()*sizeof(uint32), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_qry_idx,d_qry_idx_sample,10000*sizeof(int),hipMemcpyDeviceToDevice));//copy oringin idx to current idx for sort
  
    // CHECK(hipMemcpyAsync(d_gene_idx, h_gene_idx.data(), h_gene_idx.size()*sizeof(uint32), hipMemcpyHostToDevice,stream));
    // CHECK(hipMemcpyAsync(d_qry_idx,d_qry_idx_sample,10000*sizeof(int),hipMemcpyDeviceToDevice,stream));//copy oringin idx to current idx for sort
    // CHECK(hipStreamSynchronize(stream));

  
    // hipMemset(d_qry_line, 0, qry_width * sizeof(float));
    // hipMemset(d_qry_rank, 0, qry_width * sizeof(float));
    // hipMemcpy(d_qry_line, h_qry_line.data(), h_qry_line.size()*sizeof(float), hipMemcpyHostToDevice);
    get_device_qry_line<<< h_gene_idx.size()/1024 + 1, 1024 >>>(d_gene_idx, qry, h_gene_idx.size(), qry_width, d_qry_line);

    //check result of get_device_qry_line()
    // vector<float> tmp_qry_line;
    // tmp_qry_line.resize(h_gene_idx.size(), 0);
    // hipMemcpy(tmp_qry_line.data(), d_qry_line, h_gene_idx.size()*sizeof(float), hipMemcpyDeviceToHost);
    // cout<<tmp_qry_line.size()<<endl;
    // for (int i = 0; i < tmp_qry_line.size(); ++i)
    //     cout<<tmp_qry_line[i]<<" ";
    // cout<<endl;
    
    // rank for qry line
  
    cout<<"thrust sortLen: "<<h_gene_idx.size()<<endl;
    //rank_by_unique(d_qry_line,d_qry_idx,h_gene_idx.size());
   // getchar();
    // thrust::device_ptr<float> thrust_InOut(d_qry_line);
    sort_by_idx1(d_qry_line,d_qry_idx,h_gene_idx.size());
   // sortrank<<< (h_gene_idx.size()/2)/1024 + 1, 1024 >>>(d_qry_line,d_qry_idx,h_gene_idx.size());
    cout<<"thrust end:"<<h_gene_idx.size()<<endl;
    // vector<float> tmp_qryed;
    // vector<int> tmp_qryed_idx;
    // tmp_qryed.resize(h_gene_idx.size(), 0);
    // tmp_qryed_idx.resize(h_gene_idx.size(), 0);

    // CHECK( hipMemcpy(tmp_qryed_idx.data(), d_qry_idx, h_gene_idx.size()*sizeof(int), hipMemcpyDeviceToHost));
    // CHECK(hipMemcpy(tmp_qryed.data(),d_qry_line, h_gene_idx.size()*sizeof(float),hipMemcpyDeviceToHost));
    // for (int i = 0; i < h_gene_idx.size(); ++i)
    //     {
    //         cout<<"data"<<tmp_qryed[i]<<" ";
    //         cout<<"idx"<<tmp_qryed_idx[i]<<endl;
    //     }
    // getchar();

    // hipMalloc((void**)&unique_idx,h_gene_idx.size()*sizeof(int));
    // hipMalloc((void**)&unique_val,h_gene_idx.size()*sizeof(float));
    
    //sort_by_idx1(d_qry_line,d_qry_idx,h_gene_idx.size());
   // sort_by_idx(d_qry_line,d_qry_idx,,,h_gene_idx.size());
    //getchar();
    // thrust::device_ptr<float> thrust_InOut(d_qry_line);
    // thrust::device_ptr<int> thrust_Idx(d_qry_idx);
    // thrust::sort_by_key(dataInOut,dataInOut+h_gene_idx.size(),dataIdx);
    // thrust::device_ptr<float> d_qry_line_Thrust(d_qry_line);

    // t0=clock();

    // thrust::sort(d_qry_line_Thrust,d_qry_line_Thrust+h_gene_idx.size());
    time=(float)(t1-t0)/CLOCKS_PER_SEC;
    cout<<"sort : "<<time<<endl;

    //thrust::sort_by_key(d_qry_line_Thrust,d_qry_line_Thrust+h_gene_idx.size(),ids);
    cout<<"thrust end"<<endl;
  //  getchar();
    
   // rankdata<<< h_gene_idx.size()/1024 + 1, 1024 >>>(d_qry_line, h_gene_idx.size(), d_qry_rank);
   
    // t1=clock();
    // time=(float)(t1-t0)/CLOCKS_PER_SEC;
    // cout<<"rankdata 1st : "<<time<<endl;
    // hipMemcpy(tmp_qry_line.data(), d_qry_rank, h_gene_idx.size()*sizeof(float), hipMemcpyDeviceToHost);
    // cout<<tmp_qry_line.size()<<endl;
    // for (int i = 0; i < tmp_qry_line.size(); ++i)
    //     cout<<tmp_qry_line[i]<<" ";
    // cout<<endl;

    // get filtered cells of ref data
    // hipMemset(d_ref_lines, 0, 1000000 * sizeof(float));
    // hipMemset(d_ref_rank, 0, 1000000 * sizeof(float));
    
    vector<float> scores;
    for (auto& label : top_labels)
    {
      
 
        uint32 pos = h_ctidx[label * 2];
        uint32 len = h_ctidx[label * 2 + 1];
        
        dim3 blockDim(32, 32);
        dim3 gridDim(len/32+1, h_gene_idx.size()/32+1);
        t0=clock();
        get_device_ref_lines<<< gridDim, blockDim >>>
            (d_gene_idx, h_gene_idx.size(), d_ctids+pos, len, d_ref, ref_width, pitchref, d_ref_lines);
        t1=clock();
        time=(float)(t1-t0)/CLOCKS_PER_SEC;
        cout<<"label:"<<label<<" get_device_ref_lines : "<<time<<endl;
        // check result of get_device_ref_lines()
        // vector<float> tmp_ref_line;
        // tmp_ref_line.resize(h_gene_idx.size()*len, 0);
        // hipMemcpy(tmp_ref_line.data(), d_ref_lines, h_gene_idx.size()*len*sizeof(float), hipMemcpyDeviceToHost);
        // float max_val = 0, total_val = 0;
        // for (int i = 0; i < tmp_ref_line.size(); ++i)
        // {
        //     max_val = max(max_val, tmp_ref_line[i]);
        //     total_val += tmp_ref_line[i];
        //     if (tmp_ref_line[i] > 10)
        //         cout<<i<<","<<tmp_ref_line[i]<<" ";
        // }
        // cout<<max_val<<" "<<total_val<<endl;

        // rank for ref lines
        t0=clock();
        for (int i = 0; i < len; ++i)
        {
            //sortrank<<< (h_gene_idx.size()/2)/1024 + 1, 1024 >>>(d_qry_line,d_qry_idx,h_gene_idx.size());
          //  sortrank<<<(h_gene_idx.size()/2)/1024 + 1, 1024>>>(d_ref_lines+i*h_gene_idx.size(),d_qry_idx,h_gene_idx.size());
    //         CHECK( hipMemcpy(tmp_qryed.data(), d_ref_lines+i*h_gene_idx.size(), h_gene_idx.size()*sizeof(float), hipMemcpyDeviceToHost));
    //         CHECK( hipMemcpy(tmp_qryed_idx.data(), d_qry_idx, h_gene_idx.size()*sizeof(int), hipMemcpyDeviceToHost));
    // for (int i = 0; i < h_gene_idx.size(); ++i)
    //     {
    //         cout<<"data"<<tmp_qryed[i]<<" ";
    //         cout<<"idx"<<tmp_qryed_idx[i]<<endl;
    //     }
    // getchar();
        sort_by_idx1(d_ref_lines+i*h_gene_idx.size(),d_qry_idx,h_gene_idx.size());
           // rankdata<<< h_gene_idx.size()/1024 + 1, 1024 >>>(d_ref_lines+i*h_gene_idx.size(), h_gene_idx.size(), d_ref_rank+i*h_gene_idx.size());
        }
        t1=clock();
        time=(float)(t1-t0)/CLOCKS_PER_SEC;
        cout<<"label:"<<label<<" rankdata  len "<<len<<"time:"<<time<<endl;
        // vector<float> tmp_ref_line;
        // tmp_ref_line.resize(h_gene_idx.size()*len, 0);
        // hipMemcpy(tmp_ref_line.data(), d_ref_lines, h_gene_idx.size()*len*sizeof(float), hipMemcpyDeviceToHost);
        // float max_val = 0, total_val = 0;
        // for (int i = 0; i < tmp_ref_line.size(); ++i)
        // {
        //     max_val = max(max_val, tmp_ref_line[i]);
        //     total_val += tmp_ref_line[i];
        // }
        // cout<<max_val<<" "<<total_val<<endl;

        // spearman
        // hipMemset(d_score, 0, 1000 * sizeof(float));
        t0=clock();
        spearman<<< len/1024 + 1, 1024 >>>(d_qry_rank, d_ref_rank, h_gene_idx.size(), len, d_score);
        t1=clock();
        time=(float)(t1-t0)/CLOCKS_PER_SEC;
        cout<<"label:"<<label<<" spearman : "<<time<<endl;
        vector<float> h_score;
        h_score.resize(len, 0);
        hipMemcpy(h_score.data(), d_score, len*sizeof(float), hipMemcpyDeviceToHost);
        // cout<<"score len: "<<len<<endl;
        // if (scores.size() == 1)
        // {
        //     auto ele = std::minmax_element(h_score.begin(), h_score.end());
        //     cout<<"ele: "<<*ele.first<<" "<<*ele.second<<endl;
        // }
        float score = percentile(h_score, len, 0.8);
        // cout<<"score: "<<score<<endl;
        scores.push_back(score);
        
        // hipMemset(d_ref_lines, 0, h_gene_idx.size() * len * sizeof(float));
    }

    // for (auto& score : scores)
    //     cout<<score<<" ";
    // cout<<endl;
    t0=clock();
    auto ele = std::minmax_element(scores.begin(), scores.end());
    float thre = *ele.second - 0.05;
    vector<uint32> res;
    for (uint32 i = 0; i < scores.size(); ++i)
    {
        if (scores[i] <= *ele.first || scores[i] < thre) continue;
        else res.push_back(top_labels[i]);
    }
    if (res.empty())
        res.push_back(top_labels.front());
    t1=clock();
    time=(float)(t1-t0)/CLOCKS_PER_SEC;
    cout<<"rest etc : "<<time<<endl;
    return res;
}

vector<uint32> finetune()
{
    // process each cell
    vector<uint32> res;
    cout<<"cell num:"<<qry_height<<endl;
    clock_t startT,endT;
    float timecell;
    // for (int i = 0; i < 1; ++i)
    //for (int i = 26; i < 27; ++i)
    for (int i = 0; i < qry_height; ++i)
    {
        startT=clock();
        float* qry_head = (float*)((char*)d_qry + i * pitchqry);

        vector<uint32> top_labels;
        uint32 start = i * ct_num;
        for (int pos = 0; pos < ct_num; ++pos)
        {
            if (h_labels.at(start + pos) != 0)
                top_labels.push_back(pos);
        }

        while (top_labels.size() > 1)
        {
            cout<<"top_labels size"<<top_labels.size()<<endl;
            top_labels = finetune_round(qry_head, top_labels);
            // for (auto& label : top_labels)
            //     cout<<label<<endl;
        }
        res.push_back(top_labels.front());
        if (i % 10 == 0)
        {
            auto now = std::chrono::system_clock::now();
            std::time_t curr_time = std::chrono::system_clock::to_time_t(now);
            cout<<std::ctime(&curr_time)<<"\tprocessed "<<i<<" cells"<<endl;
        }
        endT=clock();
        timecell=(float)(endT-startT)/CLOCKS_PER_SEC;
        cout<<"cell"<<i<<"procTime:"<<timecell<<endl;
    }
 
    return res;
}
