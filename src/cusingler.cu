#include "hip/hip_runtime.h"

#include "cusingler.cuh"
#include "timer.h"

#include <stdlib.h>
#include <stdio.h>

#include <cmath>
#include <chrono>
#include <thread>
#include <iostream>
#include <set>
#include <algorithm>

#include "hip/hip_runtime.h"
#include "hip/hip_math_constants.h"

hipError_t errcode;
hipStream_t stream;
uint16* d_ref, *d_qry;
vector<float> h_labels;
uint32 ref_height, ref_width, qry_height, qry_width;
uint32 ct_num;
uint32* d_ctids;
vector<uint32> h_ctidx;
vector<uint32> h_ctdiff, h_ctdidx;
size_t pitchref;
size_t pitchqry;

uint32* d_gene_idx;
uint16 *d_ref_lines, *d_qry_line;
float *d_ref_rank, *d_qry_rank;
float *d_score;

// unit is MB
uint32 getUsedMem()
{
    size_t free, total;
    hipMemGetInfo(&free, &total);
    return (total-free)/1024/1024;
}
bool err_check()
{
    if(errcode!=hipSuccess)
        std::cout << "cudaerrcode:"<<errcode<<" line = %d" << __LINE__<<endl;
     return true;
}
bool init()
{
    stream =NULL;
    d_ref = NULL;
    d_qry = NULL;
    ref_height = ref_width = qry_height = qry_width = 0;
    ct_num = 0;
    d_ctids = NULL;
    pitchref=0;
    pitchqry=0;
    return true;
}

bool destroy()
{
    hipFree(d_ref);
    hipFree(d_qry);
    // hipFree(d_labels);
    hipFree(d_ctids);
    hipStreamDestroy(stream);
    // hipFree(d_ctidx);
    // hipFree(d_ctdiff);
    // hipFree(d_ctdidx);

    hipFree(d_gene_idx);
    hipFree(d_qry_line);
    hipFree(d_qry_rank);
    hipFree(d_ref_lines);
    hipFree(d_ref_rank);
    hipFree(d_score);

    return true;
}

bool copyin(InputData& rawdata, vector<uint32>& ctids, vector<uint32>& ctidx, vector<uint32>& ctdiff, vector<uint32>& ctdidx,
    vector<uint16>& ref, vector<uint16>& qry)
{
    ref_height = rawdata.ref_cell_num;
    ref_width = rawdata.ref_gene_num;
    qry_height = rawdata.test_cell_num;
    qry_width = rawdata.test_gene_num;
    ct_num = rawdata.celltypes.size();

    // float max_val = 0;
    // for (int i = 0; i < qry_width; ++i)
    // {
    //     max_val = max(max_val, rawdata.test[i]);
    // }
    // cout<<"qry width: "<<qry_width<<endl;
    // cout<<"qry max value: "<<max_val<<endl;
    //

    hipError_t cudaerr;
    cudaerr=hipMallocPitch((void**)&d_ref,&pitchref,ref_width*sizeof(uint16),ref_height);
    hipMallocPitch((void**)&d_qry,&pitchqry,qry_width*sizeof(uint16),qry_height);
    
    std::cout<<"pitchref: "<<pitchref<<std::endl;
    std::cout<<"pitchqry: "<<pitchqry<<std::endl;

    //hipMalloc((void**)&d_ref, ref_height * ref_width * sizeof(float));
    //hipMalloc((void**)&d_qry, qry_height * qry_width * sizeof(float));
    // hipMalloc((void**)&d_labels, qry_height * ct_num * sizeof(float));
    hipMalloc((void**)&d_ctids, ctids.size() * sizeof(uint32));
    // hipMalloc((void**)&d_ctidx, ctidx.size() * sizeof(uint32));
    // hipMalloc((void**)&d_ctdiff, ctdiff.size() * sizeof(uint32));
    // hipMalloc((void**)&d_ctdidx, ctdidx.size() * sizeof(uint32));

    hipMemcpy2DAsync(d_ref,pitchref, ref.data(), ref_width * sizeof(uint16),ref_width * sizeof(uint16),ref_height,hipMemcpyHostToDevice,stream);
    hipMemcpy2DAsync(d_qry,pitchqry, qry.data(),qry_width * sizeof(uint16),qry_width * sizeof(uint16),qry_height,hipMemcpyHostToDevice,stream);
   

    // hipMemcpyAsync(d_ref, rawdata.ref.data(), ref_height * ref_width * sizeof(float), hipMemcpyHostToDevice,stream);
    // hipMemcpyAsync(d_qry, rawdata.test.data(), qry_height * qry_width * sizeof(float), hipMemcpyHostToDevice,stream);
    // // hipMemcpy(d_labels, rawdata.labels.data(), qry_height * ct_num * sizeof(float), hipMemcpyHostToDevice);
    h_labels = rawdata.labels;
    hipMemcpyAsync(d_ctids, ctids.data(), ctids.size() * sizeof(uint32), hipMemcpyHostToDevice,stream);
    // hipMemcpy(d_ctidx, ctidx.data(), ctidx.size() * sizeof(uint32), hipMemcpyHostToDevice);
    h_ctidx = ctidx;
    // hipMemcpy(d_ctdiff, ctdiff.data(), ctdiff.size() * sizeof(uint32), hipMemcpyHostToDevice);
    // hipMemcpy(d_ctdidx, ctdidx.data(), ctdidx.size() * sizeof(uint32), hipMemcpyHostToDevice);
    h_ctdiff = ctdiff;
    h_ctdidx = ctdidx;
    hipStreamSynchronize(stream);
    // std::this_thread::sleep_for(std::chrono::seconds(5));
    std::cout<<"used gpu mem(MB): "<<getUsedMem()<<std::endl;

    hipMalloc((void**)&d_gene_idx, qry_width * sizeof(uint32));
    hipMalloc((void**)&d_qry_line, qry_width * sizeof(uint16));
    hipMalloc((void**)&d_qry_rank, qry_width * sizeof(float));
    hipMalloc((void**)&d_ref_lines, 1000000000 * sizeof(uint16));
    hipMalloc((void**)&d_ref_rank, 1000000000 * sizeof(float));
    hipMalloc((void**)&d_score, 100000 * sizeof(float));

    return true;
}

__global__ void get_device_qry_line(uint32* gene_idx, uint16* qry, const uint32 len, const uint32 gene_len, uint16* res)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < len)
    {
        res[tid] = qry[gene_len-gene_idx[tid]-1];//gene_len-1=idx_max    g_idx  int->float res dqry-line descending order
    }
}

__global__ void get_device_ref_lines(uint32* gene_idx, const uint32 gene_len,
    uint32* cell_idx, const uint32 cell_len, uint16* ref, const uint32 ref_width, 
    const uint32 ref_pitch, uint16* res)
{
    int nx = blockIdx.x * blockDim.x + threadIdx.x;
    int ny = blockIdx.y * blockDim.y + threadIdx.y;
    if (nx < cell_len && ny < gene_len)
    {
        uint16* row_head = (uint16*)((char*)ref + (uint64)(cell_idx[nx]) * ref_pitch);
        res[nx * gene_len + ny] = row_head[ref_width - gene_idx[ny] - 1];
    }
}

__global__ void rankdata(uint16* qry, const uint32 len, float* res)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < len)
    {
        float r = 1, s = 0;
        for (int i = 0; i < len; ++i)
        {
            if (qry[tid] == qry[i])
                s += 1;
            else if (qry[tid] > qry[i])
                r += 1;
        }
        res[tid] = r + float(s-1)/2;
    }
}

__global__ void rankdata_bin(uint16* qry, const uint32 cols, const uint32 rows, float* res)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < rows)
    {
        uint16* q = qry + tid*cols;
        float*  r = res + tid*cols;
        // travel for getting bin count
        uint16 bins[80] = {0};
        for (int i = 0; i < cols; ++i)
            bins[q[i]]++;

        // calculate real rank
        float ranks[80];
        float start = 0;
        for (int i = 0; i < 80; ++i)
        {
            // if (bins[i] == 0) continue;
            ranks[i] = start + (bins[i]+1)*0.5;
            start += bins[i];
        }

        // assign rank value
        for (int i = 0; i < cols; ++i)
            r[i] = ranks[q[i]];
    }
}

__global__ void spearman(float* qry, float* ref, const uint32 gene_num, const uint32 cell_num, float* score)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < cell_num)
    {
        float mean = (gene_num+1)/2.0;
        float sumxy = 0, sumxx = 0, sumyy = 0;
        for (int i = 0; i < gene_num; ++i)
        {
            float x = qry[i] - mean;
            float y = ref[tid * gene_num + i] - mean;
            sumxy += x * y;
            sumxx += x * x;
            sumyy += y * y;
        }
        float divisor = sqrt(sumxx * sumyy);
        if (divisor != 0)
            score[tid] = sumxy / divisor;
        else
            score[tid] = HIP_NAN_F;
    }
}

float percentile(vector<float> arr, int len, float p)
{
    if (len <= 1) return arr.front();

    float res;
    std::sort(arr.begin(), arr.begin()+len);

    vector<float> index;
    float step = 1.0/(len-1);
    for (int i = 0; i < len; ++i)
        index.push_back(i*step);
   
    if (p <= index.front())
    {
        res = arr[0];
    }
    else if (index.back() <= p)
    {
        res = arr[len-1];
    }
    else
    {
        auto it = lower_bound(index.begin(), index.end(), p);
        float prevIndex = *(it-1);
        float prevValue = arr.at(it - index.begin()-1);
        float nextValue = arr.at(it - index.begin());
        // linear interpolation
        res = (p - prevIndex) * (nextValue - prevValue) / step + prevValue;
    }
    return res;
}

vector<uint32> finetune_round(uint16* qry, vector<uint32> top_labels)
{
    // get filtered genes
    // cout<<"top_labels: ";
    // for (auto& label : top_labels)
    //     cout<<label<<" ";
    // cout<<"\ntop label num: "<<top_labels.size() <<" ct_num: "<<ct_num<<endl;
    set<uint32> uniq_genes;
    int gene_thre = round(500 * pow((2/3.0), log2(top_labels.size())));
    // cout<<"gene_thre: "<<gene_thre<<endl;
    
    for (auto& i : top_labels)//??line 159  topl cant be 0??
    {
        for (auto& j : top_labels)
        {
            if (i == j)//same cant be 0?
                continue;
            int pos = h_ctdidx[(i * ct_num + j) * 2];
            int len = h_ctdidx[(i * ct_num + j) * 2 + 1];
            if (len > gene_thre)
                len = gene_thre;
            uniq_genes.insert(h_ctdiff.begin()+pos, h_ctdiff.begin()+pos+len);
            // cout<<"temp uniq genes size: "<<uniq_genes.size()<<endl;
        }
    }
    // cout<<"uniq genes size: "<<uniq_genes.size()<<endl;
    
    vector<uint32> h_gene_idx(uniq_genes.begin(), uniq_genes.end());

    // transfer qry data from cpu to gpu
    hipMemcpy(d_gene_idx, h_gene_idx.data(), h_gene_idx.size()*sizeof(uint32), hipMemcpyHostToDevice);
    // hipMemset(d_qry_line, 0, qry_width * sizeof(float));
    // hipMemset(d_qry_rank, 0, qry_width * sizeof(float));
    // hipMemcpy(d_qry_line, h_qry_line.data(), h_qry_line.size()*sizeof(float), hipMemcpyHostToDevice);
    get_device_qry_line<<< h_gene_idx.size()/1024 + 1, 1024 >>>(d_gene_idx, qry, h_gene_idx.size(), qry_width, d_qry_line);

    //check result of get_device_qry_line()
    // vector<float> tmp_qry_line;
    // tmp_qry_line.resize(h_gene_idx.size(), 0);
    // hipMemcpy(tmp_qry_line.data(), d_qry_line, h_gene_idx.size()*sizeof(float), hipMemcpyDeviceToHost);
    // cout<<tmp_qry_line.size()<<endl;
    // for (int i = 0; i < tmp_qry_line.size(); ++i)
    //     cout<<tmp_qry_line[i]<<" ";
    // cout<<endl;

    // rank for qry line
    rankdata<<< h_gene_idx.size()/1024 + 1, 1024 >>>(d_qry_line, h_gene_idx.size(), d_qry_rank);
    // rankdata_bin<<< 1, 1 >>>(d_qry_line, h_gene_idx.size(), 1, d_qry_rank);

    // hipMemcpy(tmp_qry_line.data(), d_qry_rank, h_gene_idx.size()*sizeof(float), hipMemcpyDeviceToHost);
    // cout<<tmp_qry_line.size()<<endl;
    // for (int i = 0; i < tmp_qry_line.size(); ++i)
    //     cout<<tmp_qry_line[i]<<" ";
    // cout<<endl;

    // get filtered cells of ref data
    // hipMemset(d_ref_lines, 0, 1000000 * sizeof(float));
    // hipMemset(d_ref_rank, 0, 1000000 * sizeof(float));

    vector<float> scores;
    size_t total_len = 0;
    for (auto& label : top_labels)
    {
        uint32 pos = h_ctidx[label * 2];
        uint32 len = h_ctidx[label * 2 + 1];
        
        dim3 blockDim(32, 32);
        dim3 gridDim(len/32+1, h_gene_idx.size()/32+1);
        get_device_ref_lines<<< gridDim, blockDim >>>
            (d_gene_idx, h_gene_idx.size(), d_ctids+pos, len, d_ref, ref_width, pitchref, d_ref_lines + total_len*h_gene_idx.size());
        total_len += len;
    }

        // check result of get_device_ref_lines()
        // if (label > 5)
        // {
        //     vector<uint16> tmp_ref_line;
        //     tmp_ref_line.resize(h_gene_idx.size()*len, 0);
        //     hipMemcpy(tmp_ref_line.data(), d_ref_lines, h_gene_idx.size()*len*sizeof(uint16), hipMemcpyDeviceToHost);
        //     uint16 max_val = 0, total_val = 0;
        //     for (int i = 0; i < tmp_ref_line.size(); ++i)
        //     {
        //         max_val = max(max_val, tmp_ref_line[i]);
        //         total_val += tmp_ref_line[i];
        //         // if (tmp_ref_line[i] > 10)
        //         //     cout<<i<<","<<tmp_ref_line[i]<<" ";
        //     }
        //     cout<<max_val<<" "<<total_val<<endl;
        // }
        // rank for ref lines
        // for (int i = 0; i < len; ++i)
        // {
        //     rankdata<<< h_gene_idx.size()/1024 + 1, 1024 >>>(d_ref_lines+i*h_gene_idx.size(), h_gene_idx.size(), d_ref_rank+i*h_gene_idx.size());
        // }
        rankdata_bin<<< total_len/256 + 1, 256 >>>(d_ref_lines, h_gene_idx.size(), total_len, d_ref_rank);
        // cout<<"rows x cols: "<<len<<" x "<<h_gene_idx.size()<<endl;

        // vector<float> tmp_ref_line;
        // tmp_ref_line.resize(h_gene_idx.size()*len, 0);
        // hipMemcpy(tmp_ref_line.data(), d_ref_lines, h_gene_idx.size()*len*sizeof(float), hipMemcpyDeviceToHost);
        // float max_val = 0, total_val = 0;
        // for (int i = 0; i < tmp_ref_line.size(); ++i)
        // {
        //     max_val = max(max_val, tmp_ref_line[i]);
        //     total_val += tmp_ref_line[i];
        // }
        // cout<<max_val<<" "<<total_val<<endl;

        // spearman
        // hipMemset(d_score, 0, 1000 * sizeof(float));
        spearman<<< total_len/1024 + 1, 1024 >>>(d_qry_rank, d_ref_rank, h_gene_idx.size(), total_len, d_score);

        vector<float> h_score;
        h_score.resize(total_len, 0);
        hipMemcpy(h_score.data(), d_score, total_len*sizeof(float), hipMemcpyDeviceToHost);
        // cout<<"score len: "<<len<<endl;
        // if (scores.size() == 1)
        // {
        //     auto ele = std::minmax_element(h_score.begin(), h_score.end());
        //     cout<<"ele: "<<*ele.first<<" "<<*ele.second<<endl;
        // }
        // if (label == 3)
        // {
        //     for (int i = 0; i < len; ++i)
        //         cout<<h_score[i]<<" ";
        //     cout<<endl;
        // }
    uint32 start = 0;
    total_len = 0;
    for (auto& label : top_labels)
    {
        uint32 len = h_ctidx[label * 2 + 1];
        total_len += len;
        
        vector<float> tmp(h_score.begin()+start, h_score.begin()+total_len);
        float score = percentile(tmp, len, 0.8);
        // cout<<label<<" score: "<<score<<endl;
        scores.push_back(score);
        start += len;
        // hipMemset(d_ref_lines, 0, h_gene_idx.size() * len * sizeof(float));
    }

    // for (auto& score : scores)
    //     cout<<score<<" ";
    // cout<<endl;

    auto ele = std::minmax_element(scores.begin(), scores.end());
    float thre = *ele.second - 0.05;
    vector<uint32> res;
    for (uint32 i = 0; i < scores.size(); ++i)
    {
        if (scores[i] <= *ele.first || scores[i] < thre) continue;
        else res.push_back(top_labels[i]);
    }
    if (res.empty())
        res.push_back(top_labels.front());

    return res;
}

vector<uint32> finetune()
{
    Timer timer("ms");
    // process each cell
    vector<uint32> res;
    // for (int i = 0; i < 1; ++i)
    for (int i = 0; i < qry_height; ++i)
    {
        uint16* qry_head = (uint16*)((char*)d_qry + i * pitchqry);

        vector<uint32> top_labels;
        uint32 start = i * ct_num;
        for (int pos = 0; pos < ct_num; ++pos)
        {
            if (h_labels.at(start + pos) != 0)
                top_labels.push_back(pos);
        }

        while (top_labels.size() > 1)
        {
            top_labels = finetune_round(qry_head, top_labels);
            // for (auto& label : top_labels)
            //     cout<<label<<endl;
        }
        res.push_back(top_labels.front());

        if (i % 10 == 0)
        {
            auto now = std::chrono::system_clock::now();
            std::time_t curr_time = std::chrono::system_clock::to_time_t(now);
            cout<<"processed "<<i<<" cells cost time(ms): "<<timer.toc()<<endl;
        }
    }
 
    return res;
}
