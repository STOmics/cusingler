#include "hip/hip_runtime.h"

#include "cusingler.cuh"

#include <stdlib.h>
#include <stdio.h>

#include <cmath>
#include <chrono>
#include <thread>
#include <iostream>
#include <set>
#include <algorithm>

#include "hip/hip_runtime.h"
#include "hip/hip_math_constants.h"

hipError_t errcode;
hipStream_t stream;
float* d_ref, *d_qry;
vector<float> h_labels;
uint32 ref_height, ref_width, qry_height, qry_width;
uint32 ct_num;
uint32* d_ctids;
vector<uint32> h_ctidx;
vector<uint32> h_ctdiff, h_ctdidx;
size_t pitchref;
size_t pitchqry;

uint32* d_gene_idx;
float* d_qry_line, *d_qry_rank;
float* d_ref_lines, *d_ref_rank;
float *d_score;

// unit is MB
uint32 getUsedMem()
{
    size_t free, total;
    hipMemGetInfo(&free, &total);
    return (total-free)/1024/1024;
}
bool err_check()
{
    if(errcode!=hipSuccess)
        std::cout << "cudaerrcode:"<<errcode<<" line = %d" << __LINE__<<endl;
     return true;
}
bool init()
{
    stream =NULL;
    d_ref = NULL;
    d_qry = NULL;
    ref_height = ref_width = qry_height = qry_width = 0;
    ct_num = 0;
    d_ctids = NULL;
    pitchref=0;
    pitchqry=0;
    return true;
}

bool destroy()
{
    hipFree(d_ref);
    hipFree(d_qry);
    // hipFree(d_labels);
    hipFree(d_ctids);
    hipStreamDestroy(stream);
    // hipFree(d_ctidx);
    // hipFree(d_ctdiff);
    // hipFree(d_ctdidx);

    hipFree(d_gene_idx);
    hipFree(d_qry_line);
    hipFree(d_qry_rank);
    hipFree(d_ref_lines);
    hipFree(d_ref_rank);
    hipFree(d_score);

    return true;
}

bool copyin(InputData& rawdata, vector<uint32>& ctids, vector<uint32>& ctidx, vector<uint32>& ctdiff, vector<uint32>& ctdidx)
{
    

    ref_height = rawdata.ref_cell_num;
    ref_width = rawdata.ref_gene_num;
    qry_height = rawdata.test_cell_num;
    qry_width = rawdata.test_gene_num;
    ct_num = rawdata.celltypes.size();

    // float max_val = 0;
    // for (int i = 0; i < qry_width; ++i)
    // {
    //     max_val = max(max_val, rawdata.test[i]);
    // }
    // cout<<"qry width: "<<qry_width<<endl;
    // cout<<"qry max value: "<<max_val<<endl;
    //

    hipError_t cudaerr;
    cudaerr=hipMallocPitch((void**)&d_ref,&pitchref,ref_width*sizeof(float),ref_height);
    hipMallocPitch((void**)&d_qry,&pitchqry,qry_width*sizeof(float),qry_height);
    
    std::cout<<"pitchref: "<<pitchref<<std::endl;
    std::cout<<"pitchqry: "<<pitchqry<<std::endl;

    //hipMalloc((void**)&d_ref, ref_height * ref_width * sizeof(float));
    //hipMalloc((void**)&d_qry, qry_height * qry_width * sizeof(float));
    // hipMalloc((void**)&d_labels, qry_height * ct_num * sizeof(float));
    hipMalloc((void**)&d_ctids, ctids.size() * sizeof(uint32));
    // hipMalloc((void**)&d_ctidx, ctidx.size() * sizeof(uint32));
    // hipMalloc((void**)&d_ctdiff, ctdiff.size() * sizeof(uint32));
    // hipMalloc((void**)&d_ctdidx, ctdidx.size() * sizeof(uint32));

    hipMemcpy2DAsync(d_ref,pitchref, rawdata.ref.data(), ref_width * sizeof(float),ref_width * sizeof(float),ref_height,hipMemcpyHostToDevice,stream);
    hipMemcpy2DAsync(d_qry,pitchqry, rawdata.test.data(),qry_width * sizeof(float),qry_width * sizeof(float),qry_height,hipMemcpyHostToDevice,stream);
   

    // hipMemcpyAsync(d_ref, rawdata.ref.data(), ref_height * ref_width * sizeof(float), hipMemcpyHostToDevice,stream);
    // hipMemcpyAsync(d_qry, rawdata.test.data(), qry_height * qry_width * sizeof(float), hipMemcpyHostToDevice,stream);
    // // hipMemcpy(d_labels, rawdata.labels.data(), qry_height * ct_num * sizeof(float), hipMemcpyHostToDevice);
    h_labels = rawdata.labels;
    hipMemcpyAsync(d_ctids, ctids.data(), ctids.size() * sizeof(uint32), hipMemcpyHostToDevice,stream);
    // hipMemcpy(d_ctidx, ctidx.data(), ctidx.size() * sizeof(uint32), hipMemcpyHostToDevice);
    h_ctidx = ctidx;
    // hipMemcpy(d_ctdiff, ctdiff.data(), ctdiff.size() * sizeof(uint32), hipMemcpyHostToDevice);
    // hipMemcpy(d_ctdidx, ctdidx.data(), ctdidx.size() * sizeof(uint32), hipMemcpyHostToDevice);
    h_ctdiff = ctdiff;
    h_ctdidx = ctdidx;
    hipStreamSynchronize(stream);
    // std::this_thread::sleep_for(std::chrono::seconds(5));
    std::cout<<"used gpu mem(MB): "<<getUsedMem()<<std::endl;

    hipMalloc((void**)&d_gene_idx, qry_width * sizeof(uint32));
    hipMalloc((void**)&d_qry_line, qry_width * sizeof(float));
    hipMalloc((void**)&d_qry_rank, qry_width * sizeof(float));
    hipMalloc((void**)&d_ref_lines, 1000000 * sizeof(float));
    hipMalloc((void**)&d_ref_rank, 1000000 * sizeof(float));
    hipMalloc((void**)&d_score, 1000 * sizeof(float));

    return true;
}

__global__ void get_device_qry_line(uint32* gene_idx, float* qry, const uint32 len, const uint32 gene_len, float* res)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < len)
    {
        res[tid] = qry[gene_len-gene_idx[tid]-1];//gene_len-1=idx_max    g_idx  int->float res dqry-line descending order
    }
}

__global__ void get_device_ref_lines(uint32* gene_idx, const uint32 gene_len,
    uint32* cell_idx, const uint32 cell_len, float* ref, const uint32 ref_width, 
    const uint32 ref_pitch, float* res)
{
    int nx = blockIdx.x * blockDim.x + threadIdx.x;
    int ny = blockIdx.y * blockDim.y + threadIdx.y;
    if (nx < cell_len && ny < gene_len)
    {
        float* row_head = (float*)((char*)ref + cell_idx[nx] * ref_pitch);
        res[nx * gene_len + ny] = row_head[ref_width - gene_idx[ny] - 1];
    }
}

__global__ void rankdata(float* qry, const uint32 len, float* res)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < len)
    {
        int r = 1, s = 0;
        for (int i = 0; i < len; ++i)
        {
            if (qry[tid] == qry[i])
                s += 1;
            else if (qry[tid] > qry[i])
                r += 1;
        }
        res[tid] = r + float(s-1)/2;
    }
}

__global__ void spearman(float* qry, float* ref, const uint32 gene_num, const uint32 cell_num, float* score)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < cell_num)
    {
        float mean = (gene_num+1)/2.0;
        float sumxy = 0, sumxx = 0, sumyy = 0;
        for (int i = 0; i < gene_num; ++i)
        {
            float x = qry[i] - mean;
            float y = ref[tid * gene_num + i] - mean;
            sumxy += x * y;
            sumxx += x * x;
            sumyy += y * y;
        }
        float divisor = sqrt(sumxx * sumyy);
        if (divisor != 0)
            score[tid] = sumxy / divisor;
        else
            score[tid] = HIP_NAN_F;
    }
}

float percentile(vector<float> arr, int len, float p)
{
    if (len <= 1) return arr.front();

    float res;
    std::sort(arr.begin(), arr.begin()+len);

    vector<float> index;
    float step = 1.0/(len-1);
    for (int i = 0; i < len; ++i)
        index.push_back(i*step);
   
    if (p <= index.front())
    {
        res = arr[0];
    }
    else if (index.back() <= p)
    {
        res = arr[len-1];
    }
    else
    {
        auto it = lower_bound(index.begin(), index.end(), p);
        float prevIndex = *(it-1);
        float prevValue = arr.at(it - index.begin()-1);
        float nextValue = arr.at(it - index.begin());
        // linear interpolation
        res = (p - prevIndex) * (nextValue - prevValue) / step + prevValue;
    }
    return res;
}

vector<uint32> finetune_round(float* qry, vector<uint32> top_labels)
{
    // get filtered genes
    // cout<<"top_labels: ";
    // for (auto& label : top_labels)
    //     cout<<label<<" ";
    // cout<<"\ntop label num: "<<top_labels.size() <<" ct_num: "<<ct_num<<endl;
    set<uint32> uniq_genes;
    int gene_thre = round(500 * pow((2/3.0), log2(top_labels.size())));
    // cout<<"gene_thre: "<<gene_thre<<endl;
    
    for (auto& i : top_labels)//??line 159  topl cant be 0??
    {
        for (auto& j : top_labels)
        {
            if (i == j)//same cant be 0?
                continue;
            int pos = h_ctdidx[(i * ct_num + j) * 2];
            int len = h_ctdidx[(i * ct_num + j) * 2 + 1];
            if (len > gene_thre)
                len = gene_thre;
            uniq_genes.insert(h_ctdiff.begin()+pos, h_ctdiff.begin()+pos+len);
            // cout<<"temp uniq genes size: "<<uniq_genes.size()<<endl;
        }
    }
    // cout<<"uniq genes size: "<<uniq_genes.size()<<endl;
    
    vector<uint32> h_gene_idx(uniq_genes.begin(), uniq_genes.end());

    // transfer qry data from cpu to gpu
    hipMemcpy(d_gene_idx, h_gene_idx.data(), h_gene_idx.size()*sizeof(uint32), hipMemcpyHostToDevice);
    // hipMemset(d_qry_line, 0, qry_width * sizeof(float));
    // hipMemset(d_qry_rank, 0, qry_width * sizeof(float));
    // hipMemcpy(d_qry_line, h_qry_line.data(), h_qry_line.size()*sizeof(float), hipMemcpyHostToDevice);
    get_device_qry_line<<< h_gene_idx.size()/1024 + 1, 1024 >>>(d_gene_idx, qry, h_gene_idx.size(), qry_width, d_qry_line);

    //check result of get_device_qry_line()
    // vector<float> tmp_qry_line;
    // tmp_qry_line.resize(h_gene_idx.size(), 0);
    // hipMemcpy(tmp_qry_line.data(), d_qry_line, h_gene_idx.size()*sizeof(float), hipMemcpyDeviceToHost);
    // cout<<tmp_qry_line.size()<<endl;
    // for (int i = 0; i < tmp_qry_line.size(); ++i)
    //     cout<<tmp_qry_line[i]<<" ";
    // cout<<endl;

    // rank for qry line
    rankdata<<< h_gene_idx.size()/1024 + 1, 1024 >>>(d_qry_line, h_gene_idx.size(), d_qry_rank);
    // hipMemcpy(tmp_qry_line.data(), d_qry_rank, h_gene_idx.size()*sizeof(float), hipMemcpyDeviceToHost);
    // cout<<tmp_qry_line.size()<<endl;
    // for (int i = 0; i < tmp_qry_line.size(); ++i)
    //     cout<<tmp_qry_line[i]<<" ";
    // cout<<endl;

    // get filtered cells of ref data
    // hipMemset(d_ref_lines, 0, 1000000 * sizeof(float));
    // hipMemset(d_ref_rank, 0, 1000000 * sizeof(float));

    vector<float> scores;
    for (auto& label : top_labels)
    {
        uint32 pos = h_ctidx[label * 2];
        uint32 len = h_ctidx[label * 2 + 1];
        
        dim3 blockDim(32, 32);
        dim3 gridDim(len/32+1, h_gene_idx.size()/32+1);
        get_device_ref_lines<<< gridDim, blockDim >>>
            (d_gene_idx, h_gene_idx.size(), d_ctids+pos, len, d_ref, ref_width, pitchref, d_ref_lines);

        // check result of get_device_ref_lines()
        // vector<float> tmp_ref_line;
        // tmp_ref_line.resize(h_gene_idx.size()*len, 0);
        // hipMemcpy(tmp_ref_line.data(), d_ref_lines, h_gene_idx.size()*len*sizeof(float), hipMemcpyDeviceToHost);
        // float max_val = 0, total_val = 0;
        // for (int i = 0; i < tmp_ref_line.size(); ++i)
        // {
        //     max_val = max(max_val, tmp_ref_line[i]);
        //     total_val += tmp_ref_line[i];
        //     if (tmp_ref_line[i] > 10)
        //         cout<<i<<","<<tmp_ref_line[i]<<" ";
        // }
        // cout<<max_val<<" "<<total_val<<endl;

        // rank for ref lines
        for (int i = 0; i < len; ++i)
        {
            rankdata<<< h_gene_idx.size()/1024 + 1, 1024 >>>(d_ref_lines+i*h_gene_idx.size(), h_gene_idx.size(), d_ref_rank+i*h_gene_idx.size());
        }

        // spearman
        // hipMemset(d_score, 0, 1000 * sizeof(float));
        spearman<<< len/1024 + 1, 1024 >>>(d_qry_rank, d_ref_rank, h_gene_idx.size(), len, d_score);

        vector<float> h_score;
        h_score.resize(len, 0);
        hipMemcpy(h_score.data(), d_score, len*sizeof(float), hipMemcpyDeviceToHost);
        float score = percentile(h_score, len, 0.8);
        // cout<<"score: "<<score<<endl;
        scores.push_back(score);
        
        // hipMemset(d_ref_lines, 0, h_gene_idx.size() * len * sizeof(float));
    }

    auto ele = std::minmax_element(scores.begin(), scores.end());
    float thre = *ele.second - 0.05;
    vector<uint32> res;
    for (uint32 i = 0; i < scores.size(); ++i)
    {
        if (scores[i] <= *ele.first || scores[i] < thre) continue;
        else res.push_back(top_labels[i]);
    }
    if (res.empty())
        res.push_back(top_labels.front());

    return res;
}

vector<uint32> finetune()
{
    // process each cell
    vector<uint32> res;
    for (int i = 0; i < qry_height; ++i)
    {
        float* qry_head = (float*)((char*)d_qry + i * pitchqry);

        vector<uint32> top_labels;
        uint32 start = i * ct_num;
        for (int pos = 0; pos < ct_num; ++pos)
        {
            if (h_labels.at(start + pos) != 0)
                top_labels.push_back(pos);
        }

        while (top_labels.size() > 1)
        {
            top_labels = finetune_round(qry_head, top_labels);
            // for (auto& label : top_labels)
            //     cout<<label<<endl;
        }
        res.push_back(top_labels.front());
    }
 
    return res;
}
